#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void hello_kernel(){
    printf("Hello World from CUDA kernel!\n");
}

int main()
{
    hello_kernel <<<1,25>>> ();
    hipDeviceSynchronize();

    hipDeviceReset();
    return 0;
}


